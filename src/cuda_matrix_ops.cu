#include "hip/hip_runtime.h"
#include "matrix.hpp"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void cudaMatrixMultiplyKernel(const float* A, const float* B, float* C, int m, int n, int p) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // m
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // p

    if (row < m && col < p) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k) {
            sum += A[row * n + k] * B[k * p + col];
        }
        C[row * p + col] = sum;
    }
}

void cudaMatrixMultiply(const Matrix& A, const Matrix& B, Matrix& C) {
    int m = A.rows;
    int n = A.cols;
    int p = B.cols;

    size_t size_A = m * n * sizeof(float);
    size_t size_B = n * p * sizeof(float);
    size_t size_C = m * p * sizeof(float);

    float *d_A, *d_B, *d_C;

    // Allocate device memory
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    // Copy data to device
    hipMemcpy(d_A, A.data.data(), size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data.data(), size_B, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 dimBlock(16, 16);
    dim3 dimGrid((p + dimBlock.x - 1) / dimBlock.x, (m + dimBlock.y - 1) / dimBlock.y);

    // Launch kernel
    cudaMatrixMultiplyKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, m, n, p);

    // Copy result back to host
    hipMemcpy(C.data.data(), d_C, size_C, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// Implement similar functions for cudaMatrixAdd and cudaMatrixTranspose
__global__ void cudaMatrixVectorMultiplyKernel(const float* A, const float* x, float* y, int m, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k) {
            sum += A[row * n + k] * x[k];
        }
        y[row] = sum;
    }
}

void cudaMatrixVectorMultiply(const Matrix& A, const std::vector<float>& x, std::vector<float>& y) {
    int m = A.rows;
    int n = A.cols;
    size_t size_A = m * n * sizeof(float);
    size_t size_x = n * sizeof(float);
    size_t size_y = m * sizeof(float);

    float *d_A, *d_x, *d_y;

    // Allocate device memory
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_x, size_x);
    hipMalloc(&d_y, size_y);

    // Copy data to device
    hipMemcpy(d_A, A.data.data(), size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_x, x.data(), size_x, hipMemcpyHostToDevice);

    // Define block and grid sizes
    int blockSize = 256;
    int gridSize = (m + blockSize - 1) / blockSize;

    // Launch kernel
    cudaMatrixVectorMultiplyKernel<<<gridSize, blockSize>>>(d_A, d_x, d_y, m, n);

    // Copy result back to host
    y.resize(m);
    hipMemcpy(y.data(), d_y, size_y, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
}

// CUDA kernel for matrix addition
__global__ void matrixAddKernel(const float* A, const float* B, float* C, int rows, int cols) {
    // Calculate the global thread ID
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Row index
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Column index

    // Compute the index in the 1D array
    int idx = row * cols + col;

    // Perform the addition if within bounds
    if (row < rows && col < cols) {
        C[idx] = A[idx] + B[idx];
    }
}


void cudaMatrixAdd(const Matrix& A, const Matrix& B, Matrix& C) {
    // Check if matrix dimensions match
    if (A.rows != B.rows || A.cols != B.cols) {
        std::cerr << "Error: Matrices A and B must have the same dimensions." << std::endl;
        return;
    }

    if (A.rows != C.rows || A.cols != C.cols) {
        std::cerr << "Error: Matrix C must have the same dimensions as A and B." << std::endl;
        return;
    }

    int rows = A.rows;
    int cols = A.cols;
    size_t size = rows * cols * sizeof(float);

    // Device pointers
    float* d_A = nullptr;
    float* d_B = nullptr;
    float* d_C = nullptr;

    hipError_t err;

    // Allocate device memory for A
    err = hipMalloc((void**)&d_A, size);
    if (err != hipSuccess) {
        std::cerr << "Error allocating device memory for A: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // Allocate device memory for B
    err = hipMalloc((void**)&d_B, size);
    if (err != hipSuccess) {
        std::cerr << "Error allocating device memory for B: " << hipGetErrorString(err) << std::endl;
        hipFree(d_A); // Free previously allocated memory
        return;
    }

    // Allocate device memory for C
    err = hipMalloc((void**)&d_C, size);
    if (err != hipSuccess) {
        std::cerr << "Error allocating device memory for C: " << hipGetErrorString(err) << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        return;
    }

    // Copy data from host to device for A
    err = hipMemcpy(d_A, A.data.data(), size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Error copying data from host to device for A: " << hipGetErrorString(err) << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return;
    }

    // Copy data from host to device for B
    err = hipMemcpy(d_B, B.data.data(), size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Error copying data from host to device for B: " << hipGetErrorString(err) << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return;
    }

    // Define block and grid sizes
    const int TILE_SIZE = 16;
    dim3 blockSize(TILE_SIZE, TILE_SIZE);
    dim3 gridSize((cols + TILE_SIZE - 1) / TILE_SIZE, (rows + TILE_SIZE - 1) / TILE_SIZE);

    // Launch the kernel
    matrixAddKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, rows, cols);

    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Error launching matrixAddKernel: " << hipGetErrorString(err) << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return;
    }

    // Copy result from device to host
    err = hipMemcpy(C.data.data(), d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "Error copying data from device to host for C: " << hipGetErrorString(err) << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return;
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

__global__ void cudaMatrixTransposeKernel(const float* A, float* B, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        B[col * m + row] = A[row * n + col];
    }
}

void cudaMatrixTranspose(const Matrix& A, Matrix& B) {
    int m = A.rows;
    int n = A.cols;
    size_t size_A = m * n * sizeof(float);
    size_t size_B = n * m * sizeof(float);

    float *d_A, *d_B;

    // Allocate device memory
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);

    // Copy data to device
    hipMemcpy(d_A, A.data.data(), size_A, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 dimBlock(16, 16);
    dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x, (m + dimBlock.y - 1) / dimBlock.y);

    // Launch kernel
    cudaMatrixTransposeKernel<<<dimGrid, dimBlock>>>(d_A, d_B, m, n);

    hipMemcpy(B.data.data(), d_B, size_B, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
}